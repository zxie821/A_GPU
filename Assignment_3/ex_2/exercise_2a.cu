
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

__host__ __device__ float3 operator+(const float3 &a, const float3 &b)
{
  return {a.x+b.x, a.y+b.y, a.z+b.z};
}
__host__ __device__ float3 operator-(const float3 &a, const float3 &b)
{
  return {a.x-b.x, a.y-b.y, a.z-b.z};
}

__host__ __device__ void update(float3 &p, float3 &v, const int it)
{
  p = p + v;
  v = v + make_float3(0.1f*it, 0.1f*it, 0.1f*it);
}

__global__ void simKernel(float3 *particles, float3 *velocities, int iterations)
{
  const int id = blockIdx.x*blockDim.x + threadIdx.x;
  for(size_t it = 0; it<iterations;++it)
  {
    update(particles[id], velocities[id], it);
  }
}


int main()
{
  int numParticles, numIterations, blockSize;
  std::cin >> numParticles >> numIterations >>blockSize;
  int byteSize = numParticles * sizeof(float3);

  //GPU SIMULATION:
  auto start_gpu = std::chrono::high_resolution_clock::now();

  float3 *gpu_particles;
  float3 *gpu_velocities;
  hipHostMalloc(&gpu_particles, byteSize, hipHostMallocDefault);
  hipHostMalloc(&gpu_velocities, byteSize, hipHostMallocDefault);

  for(size_t i=0; i<numParticles; ++i)
  {
    gpu_particles[i] = make_float3(.1f,.1f,.1f);
    gpu_velocities[i] = make_float3(.01f,.01f,.01f);
  }

  float3 *dgpu_particles;
  float3 *dgpu_velocities;
  hipMalloc(&dgpu_particles, byteSize);
  hipMalloc(&dgpu_velocities, byteSize);
  for(int i=0; i<numIterations; ++i)
  {
    hipMemcpy(dgpu_particles, gpu_particles, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(dgpu_velocities, gpu_velocities, byteSize, hipMemcpyHostToDevice);

    simKernel<<<(numParticles+blockSize-1)/blockSize, blockSize>>>(dgpu_particles, 
      dgpu_velocities, 1);
    hipDeviceSynchronize();
    hipMemcpy(gpu_particles, dgpu_particles, byteSize, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_velocities, dgpu_velocities, byteSize, hipMemcpyDeviceToHost);
  }

  hipFree(dgpu_particles);
  hipFree(dgpu_velocities);
  hipHostFree(gpu_particles);
  hipHostFree(gpu_velocities);

  auto end_gpu = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> time_gpu = end_gpu-start_gpu;
  std::cout<<"GPU simulation time:"<<time_gpu.count()<<std::endl;

  return 0;
}