
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

__device__ bool checkCircle(float x, float y)
{
  if(sqrt((x*x) + (y*y)) <= 1.0)
  {
    return true;
  }
  return false;
}

__global__ void piKernel(int *d_res, int iterations, int totalIterations, hiprandState *states)
{
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id >= totalIterations)
    return;
  int localCount = 0;
  const int seed = id;
  hiprand_init(seed, id, 0, &states[id]);
  for(size_t it=0; it<iterations; ++it)
  {
    float x = hiprand_uniform(&states[id]);
    float y = hiprand_uniform(&states[id]);
    if(checkCircle(x,y))
    {
      ++localCount;
    }
  }
  atomicAdd(d_res, localCount);
}
int main()
{
  int blockSize, iterationPerCThread, totalIterations;
  std::cin >>blockSize >>iterationPerCThread >> totalIterations;

  auto start_gpu = std::chrono::high_resolution_clock::now();
  int counter = 0;
  int *dCounter=0;
  hipMalloc(&dCounter, sizeof(int));
  hipMemset(dCounter, 0, sizeof(int));

  hiprandState *dev_random;
  int numThread = totalIterations / iterationPerCThread;
  int numBlock = (numThread+blockSize-1)/blockSize;
  hipMalloc((void**)&dev_random, numBlock*blockSize*sizeof(hiprandState));

  piKernel<<<numBlock, blockSize>>>(dCounter, iterationPerCThread,
    totalIterations, dev_random);
  hipDeviceSynchronize();

  hipMemcpy(&counter, dCounter, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dCounter);

  double pi = ((double)counter / (double)totalIterations) * 4.0;  
  float pi_f = ((float)counter / (float)totalIterations) * 4.0;  
  auto end_gpu = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> time_gpu = end_gpu-start_gpu;

  std::cout<<"GPU simulation time:"<<time_gpu.count()<<std::endl;
  std::cout<<"Pi result is:"<<pi<<std::endl;
  std::cout<<"float Pi result is:"<<pi_f<<std::endl;
  std::cout<<"difference between float and double Pi is:"<<pi_f-pi<<std::endl;
  return 0;
}